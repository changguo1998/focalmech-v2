#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include "types.h"
#include "GlobalSetting.h"
#include "Record.h"
#include "GreenFunction.h"
#include "Phase.h"
#include "Result.h"
#include "misfits.h"
#include "kernel.h"

// #define DEBUG

#define sind(x) sin((x) * M_PI / 180.0)
#define cosd(x) cos((x) * M_PI / 180.0)

__device__ void dc2ts(Float64 *mt11, Float64 *mt22, Float64 *mt33, Float64 *mt12, Float64 *mt13, Float64 *mt23,
                      Float64 s, Float64 d, Float64 r)
{
    Float64 ss = sind(s), cs = cosd(s), s2s = sind(2.0 * s), c2s = cosd(2.0 * s);
    Float64 sd = sind(d), cd = cosd(d), s2d = sind(2.0 * d), c2d = cosd(2.0 * d);
    Float64 sr = sind(r), cr = cosd(r), s2r = sind(2.0 * r), c2r = cosd(2.0 * r);

    *mt11 = -(s2s * sd * cr + (ss * ss) * s2d * sr);
    *mt22 = (s2s * sd * cr - (cs * cs) * s2d * sr);
    *mt33 = s2d * sr;
    *mt12 = (c2s * sd * cr + 0.5 * s2s * s2d * sr);
    *mt13 = -(cs * cd * cr + ss * c2d * sr);
    *mt23 = -(ss * cd * cr - cs * c2d * sr);
    return;
}

void dc2ts_omp(Float64 *mt11, Float64 *mt22, Float64 *mt33, Float64 *mt12, Float64 *mt13, Float64 *mt23,
               Float64 s, Float64 d, Float64 r)
{
    Float64 ss = sind(s), cs = cosd(s), s2s = sind(2.0 * s), c2s = cosd(2.0 * s);
    Float64 sd = sind(d), cd = cosd(d), s2d = sind(2.0 * d), c2d = cosd(2.0 * d);
    Float64 sr = sind(r), cr = cosd(r), s2r = sind(2.0 * r), c2r = cosd(2.0 * r);

    *mt11 = -(s2s * sd * cr + (ss * ss) * s2d * sr);
    *mt22 = (s2s * sd * cr - (cs * cs) * s2d * sr);
    *mt33 = s2d * sr;
    *mt12 = (c2s * sd * cr + 0.5 * s2s * s2d * sr);
    *mt13 = -(cs * cd * cr + ss * c2d * sr);
    *mt23 = -(ss * cd * cr - cs * c2d * sr);
    return;
}

#define lin2cart(v, n) \
    do                 \
    {                  \
        v = res % (n); \
        res /= (n);    \
    } while (0)

__global__ void kernel_gpu(GlobalSetting *gs, Record *rs, GreenFunction *gfs, Phase *ps,
                           Float64 *mis_waveform, Int64 *mis_shift, Float64 *mis_pol, Float64 *mis_psr)
{
    Int64 idx, ifreq, iphase, istrike, idip, irake, res;

    idx = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx >= (gs->n_frequency_pair * gs->n_phase * gs->nstrike * gs->ndip * gs->nrake))
    {
        // printf("(kernel_gpu) out of bounds %lld, nfreq: %lld, nphase: %lld\n", idx, gs->n_frequency_pair, gs->n_phase);
        return;
    }

    res = idx;
    lin2cart(ifreq, gs->n_frequency_pair);
    lin2cart(iphase, gs->n_phase);
    lin2cart(istrike, gs->nstrike);
    lin2cart(idip, gs->ndip);
    lin2cart(irake, gs->nrake);

#ifdef DEBUG
    // printf("(kernel_gpu) idx: %lld, ifreq: %lld, iphase: %lld, istrike: %lld, idip: %lld, irake: %lld, res: %lld\n",
    //        idx, ifreq, iphase, istrike, idip, irake, res);
#endif

    mis_waveform[idx] = 12345.0;
    mis_shift[idx] = 12345;
    mis_pol[idx] = 12345.0;
    mis_psr[idx] = 12345.0;

    Float64 strike, dip, rake, mt11, mt22, mt33, mt12, mt13, mt23;

    strike = istrike * gs->dstrike;
    dip = idip * gs->ddip;
    rake = irake * gs->drake - 90.0;

    dc2ts(&mt11, &mt22, &mt33, &mt12, &mt13, &mt23, strike, dip, rake);

#ifdef DEBUG
    // printf("(kernel_gpu) strike: %.1lf, dip: %.1lf, rake: %.1lf, mt: [%.1lf,%.1lf,%.1lf,%.1lf,%.1lf,%.1lf,]\n",
    //        strike, dip, rake, mt11, mt22, mt33, mt12, mt13, mt23);
#endif

    Int64 igf, irec, ngf;
    for (irec = 0; irec < gs->n_record; irec++)
    {
#ifdef DEBUG
        printf("(kernel_gpu) phase.rid: %lld, r.id: %lld\n", ps[iphase].rid, rs[irec].id);
#endif
        if (rs[irec].id == ps[iphase].rid)
            break;
    }

    if (irec >= gs->n_record)
        return;

    ngf = gs->n_record * gs->n_event_location;
    for (igf = 0; igf < ngf; igf++)
    {
#ifdef DEBUG
        printf("(kernel_gpu) phase.rid: %lld, phase.eid: %lld, gf.rid: %lld, gf.eid: %lld\n",
               ps[iphase].rid, ps[iphase].eid, gfs[igf].rid, gfs[igf].eid);
#endif
        if ((gfs[igf].rid == ps[iphase].rid) && (gfs[igf].eid == ps[iphase].eid))
            break;
    }

    if (igf >= ngf)
        return;

#ifdef DEBUG
    printf("(kernel_gpu) irec: %lld, igf: %lld\n", irec, igf);
#endif

    Float64 nxc;
    Int64 datshift;
    datshift = ifreq * rs[irec].npts;
    maximum_xcorr_mt_gpu(&nxc, &mis_shift[idx], ps[iphase].length, 20,
                         rs[irec].npts, ps[iphase].Rstart, &(rs[irec].data[datshift]),
                         rs[irec].npts, ps[iphase].Estart,
                         &(gfs[igf].g11[datshift]), &(gfs[igf].g22[datshift]), &(gfs[igf].g33[datshift]),
                         &(gfs[igf].g12[datshift]), &(gfs[igf].g13[datshift]), &(gfs[igf].g23[datshift]),
                         mt11, mt22, mt33, mt12, mt13, mt23);
    mis_waveform[idx] = sqrt((1.0 - nxc) * 0.5);
    mis_pol[idx] = 0.0;
    mis_psr[idx] = 0.0;
    return;
}

void kernel_omp(Int64 idx, GlobalSetting *gs, Record *rs, GreenFunction *gfs, Phase *ps,
                Float64 *mis_waveform, Int64 *mis_shift, Float64 *mis_pol, Float64 *mis_psr)
{
    // Int64 idx;
    Int64 ifreq, iphase, istrike, idip, irake, res;

    // idx = threadIdx.x + (blockIdx.x * blockDim.x);

    res = idx;
    lin2cart(ifreq, gs->n_frequency_pair);
    lin2cart(iphase, gs->n_phase);
    lin2cart(istrike, gs->nstrike);
    lin2cart(idip, gs->ndip);
    lin2cart(irake, gs->nrake);

#ifdef DEBUG
    // printf("(kernel_omp) idx: %lld, ifreq: %lld, iphase: %lld, istrike: %lld, idip: %lld, irake: %lld, res: %lld\n",
    //        idx, ifreq, iphase, istrike, idip, irake, res);
#endif

    mis_waveform[idx] = 12345.0;
    mis_shift[idx] = 12345;
    mis_pol[idx] = 12345.0;
    mis_psr[idx] = 12345.0;

    Float64 strike, dip, rake, mt11, mt22, mt33, mt12, mt13, mt23;

    strike = istrike * gs->dstrike;
    dip = idip * gs->ddip;
    rake = irake * gs->drake - 90.0;

    dc2ts_omp(&mt11, &mt22, &mt33, &mt12, &mt13, &mt23, strike, dip, rake);

#ifdef DEBUG
    // printf("(kernel_omp) strike: %.1lf, dip: %.1lf, rake: %.1lf, mt: [%.1lf,%.1lf,%.1lf,%.1lf,%.1lf,%.1lf,]\n",
    //        strike, dip, rake, mt11, mt22, mt33, mt12, mt13, mt23);
#endif

    Int64 igf, irec, ngf;
    for (irec = 0; irec < gs->n_record; irec++)
    {
#ifdef DEBUG
        printf("(kernel_omp) phase.rid: %lld, r.id: %lld\n", ps[iphase].rid, rs[irec].id);
#endif
        if (rs[irec].id == ps[iphase].rid)
            break;
    }

    if (irec >= gs->n_record)
        return;

    ngf = gs->n_record * gs->n_event_location;
    for (igf = 0; igf < ngf; igf++)
    {
#ifdef DEBUG
        printf("(kernel_omp) phase.rid: %lld, phase.eid: %lld, gf.rid: %lld, gf.eid: %lld\n",
               ps[iphase].rid, ps[iphase].eid, gfs[igf].rid, gfs[igf].eid);
#endif
        if ((gfs[igf].rid == ps[iphase].rid) && (gfs[igf].eid == ps[iphase].eid))
            break;
    }

    if (igf >= ngf)
        return;

#ifdef DEBUG
    printf("(kernel_omp) irec: %lld, igf: %lld\n", irec, igf);
#endif

    Float64 nxc;
    Int64 datshift;
    datshift = ifreq * rs[irec].npts;
    maximum_xcorr_mt_cpu(&nxc, &mis_shift[idx], ps[iphase].length, 10,
                         rs[irec].npts, ps[iphase].Rstart, &(rs[irec].data[datshift]),
                         rs[irec].npts, ps[iphase].Estart,
                         &(gfs[igf].g11[datshift]), &(gfs[igf].g22[datshift]), &(gfs[igf].g33[datshift]),
                         &(gfs[igf].g12[datshift]), &(gfs[igf].g13[datshift]), &(gfs[igf].g23[datshift]),
                         mt11, mt22, mt33, mt12, mt13, mt23);
    mis_waveform[idx] = sqrt((1.0 - nxc) * 0.5);
    mis_pol[idx] = 0.0;
    mis_psr[idx] = 0.0;
    return;
}

static inline Int64 _max_(Int64 a, Int64 b) { return (a > b) ? a : b; }

void call_kernel_gpu(GlobalSetting_xPU *gs, Record_xPU *rlist, GreenFunction_xPU *gflist, Phase_xPU *plist, Result_xPU *result)
{
    Int64 n_threads;
    n_threads = gs->cpu->n_frequency_pair * gs->cpu->n_phase * gs->cpu->nstrike * gs->cpu->ndip * gs->cpu->nrake;
    int block = 32;
    int grid = (n_threads - 1) / 32 + 1;
    Result_xPU_sync(result);
    kernel_gpu<<<grid, block>>>(gs->gpu, rlist->gpu, gflist->gpu, plist->gpu,
                                result->waveform_gpu, result->shift_gpu, result->polarity_gpu, result->ps_ratio_gpu);
    hipDeviceSynchronize();
    result->mgpu = _max_(result->mcpu, result->mgpu) + 1;
    Result_xPU_sync(result);
}

void call_kernel_omp(GlobalSetting_xPU *gs, Record_xPU *rlist, GreenFunction_xPU *gflist, Phase_xPU *plist, Result_xPU *result)
{
    Int64 n_threads, idx;
    n_threads = gs->cpu->n_frequency_pair * gs->cpu->n_phase * gs->cpu->nstrike * gs->cpu->ndip * gs->cpu->nrake;
    Result_xPU_sync(result);
    omp_set_num_threads(12);

#pragma omp parallel for num_threads(12) default(none) shared(n_threads, gs, rlist, gflist, plist, result) private(idx)
    {
        for (idx = 0; idx < n_threads; idx++)
            kernel_omp(idx, gs->cpu, rlist->cpu, gflist->cpu, plist->cpu,
                       result->waveform, result->shift, result->polarity, result->ps_ratio);
    }

    result->mcpu = _max_(result->mcpu, result->mgpu) + 1;
    Result_xPU_sync(result);
}
