#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "types.h"
#include "GlobalSetting.h"
#include "Record.h"
#include "GreenFunction.h"
#include "Phase.h"
#include "Result.h"
#include "misfits.h"
#include "kernel.h"
#include "io.h"

// #define GPU
#define DEBUG

#define WRITE_LOG(...)                \
    do                                \
    {                                 \
        printf(__VA_ARGS__);          \
        fprintf(fp_log, __VA_ARGS__); \
    } while (0)

static inline Int64 _max_(Int64 a, Int64 b) { return (a > b) ? a : b; }

int main(int argc, char *argv[])
{
    FILE *fp_input, *fp_log, *fp_output;
    Int64 n_freq, n_phase, n_fm;

    GlobalSetting_xPU global_setting;
    Record_xPU record;
    GreenFunction_xPU greenfunction_database;
    Phase_xPU phase_list;
    Result_xPU result_buffer;

    fp_input = fopen("input_db.bin", "rb");
    if (fp_input == NULL)
    {
        printf("Error opening file input\n");
        exit(1);
    }

    fp_log = fopen("log.txt", "w");
    if (fp_log == NULL)
    {
        printf("Error opening file log\n");
        exit(-1);
    }

    WRITE_LOG("Load input database\n");
    load_database(&global_setting, &record, &greenfunction_database, &phase_list, fp_input);
    fclose(fp_input);

    WRITE_LOG("Allocate result buffers\n");
    n_freq = global_setting.cpu->n_frequency_pair;
    n_phase = global_setting.cpu->n_phase;
    n_fm = global_setting.cpu->nstrike * global_setting.cpu->ndip * global_setting.cpu->nrake;
#ifdef DEBUG
    printf("n_freq: %lld, n_phase: %lld, nfm :% lld\n", n_freq, n_phase, n_fm);
#endif
    Result_xPU_alloc(&result_buffer, n_freq, n_phase, n_fm);
    Int64 nresult = n_freq * n_phase * n_fm;
    memset(result_buffer.waveform, 0, nresult * sizeof(Float64));
    memset(result_buffer.shift, 0, nresult * sizeof(Int64));
    memset(result_buffer.polarity, 0, nresult * sizeof(Float64));
    memset(result_buffer.ps_ratio, 0, nresult * sizeof(Float64));
    result_buffer.mcpu = 1;
    result_buffer.mgpu = 0;
    Result_xPU_sync(&result_buffer);
    Int64 i;
    for (i = 0; i < nresult; i++)
    {
        result_buffer.waveform[i] = 10.0;
        result_buffer.shift[i] = 10;
        result_buffer.polarity[i] = 10.0;
        result_buffer.ps_ratio[i] = 10.0;
    }

    WRITE_LOG("Start kernel\n");
#ifdef GPU
    call_kernel_gpu(&global_setting, &record, &greenfunction_database, &phase_list, &result_buffer);
#else
    call_kernel_omp(&global_setting, &record, &greenfunction_database, &phase_list, &result_buffer);
#endif

    WRITE_LOG("Write result to file\n");
#ifdef GPU
    fp_output = fopen("result_gpu.bin", "wb");
#else
    fp_output = fopen("result_omp.bin", "wb");
#endif
    if (fp_output == NULL)
    {
        printf("Error opening output\n");
        exit(-1);
    }
    Result_xPU_write(&result_buffer, fp_output);
    fclose(fp_output);

    WRITE_LOG("Free memory\n");
    Result_xPU_free(&result_buffer);
    Phase_xPU_free(&phase_list);
    GreenFunction_xPU_free(&greenfunction_database);
    Record_xPU_free(&record);
    GlobalSetting_xPU_free(&global_setting);
    WRITE_LOG("End of program\n");
    fclose(fp_log);
    printf("Done\n");

    return 0;
}
