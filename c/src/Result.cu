
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "types.h"
#include "Result.h"

// #define DEBUG

void Result_xPU_alloc(Result_xPU *res, Int64 n_freq, Int64 n_phase, Int64 n_fm)
{
    Int64 n = n_freq * n_phase * n_fm;
    res->n_freq = n_freq;
    res->n_phase = n_phase;
    res->n_fm = n_fm;
    res->mcpu = 0;
    res->mgpu = 0;
    res->waveform = (Float64 *)malloc(n * sizeof(Float64));
    res->shift = (Int64 *)malloc(n * sizeof(Int64));
    res->polarity = (Float64 *)malloc(n * sizeof(Float64));
    res->ps_ratio = (Float64 *)malloc(n * sizeof(Float64));
    hipMalloc(&(res->waveform_gpu), n * sizeof(Float64));
    hipMalloc(&(res->shift_gpu), n * sizeof(Int64));
    hipMalloc(&(res->polarity_gpu), n * sizeof(Float64));
    hipMalloc(&(res->ps_ratio_gpu), n * sizeof(Float64));
}

#define _R_FREE_MAT(var)                \
    if (res->##var)                     \
        do                              \
        {                               \
            free(res->##var);           \
            res->##var = NULL;          \
        } while (0);                    \
    if (res->##var##_gpu)               \
        do                              \
        {                               \
            hipFree(res->##var##_gpu); \
            res->##var##_gpu = NULL;    \
    } while (0)

void Result_xPU_free(Result_xPU *res)
{
    _R_FREE_MAT(waveform);
    _R_FREE_MAT(shift);
    _R_FREE_MAT(polarity);
    _R_FREE_MAT(ps_ratio);
}

#undef _R_FREE_MAT

Int64 _max_(Int64 a, Int64 b) { return (a > b) ? a : b; }

void Result_xPU_sync(Result_xPU *res)
{
    hipDeviceSynchronize();
    Int64 n;
#ifdef DEBUG
    Float64 fbuf;
    printf("(Result_xPU_sync) start\n");
#endif
    n = res->n_freq * res->n_phase * res->n_fm;
#ifdef DEBUG
    printf("(Result_xPU_sync) n: %lld\n", n);
#endif
    if (res->mcpu > res->mgpu)
    {
#ifdef DEBUG
        printf("(Result_xPU_sync) CPU(%lld)->GPU(%lld)\n", res->mcpu, res->mgpu);
#endif
        hipMemcpy(res->waveform_gpu, res->waveform, n * sizeof(Float64), hipMemcpyHostToDevice);
        hipMemcpy(res->shift_gpu, res->shift, n * sizeof(Int64), hipMemcpyHostToDevice);
        hipMemcpy(res->polarity_gpu, res->polarity, n * sizeof(Float64), hipMemcpyHostToDevice);
        hipMemcpy(res->ps_ratio_gpu, res->ps_ratio, n * sizeof(Float64), hipMemcpyHostToDevice);
#ifdef DEBUG
        hipMemcpy(&fbuf, res->waveform_gpu, sizeof(Float64), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("(Result_xPU_sync) waveform[0] cpu: %lf, gpu: %lf\n", res->waveform[0], fbuf);
#endif
    }
    if (res->mcpu < res->mgpu)
    {
#ifdef DEBUG
        printf("(Result_xPU_sync) GPU(%lld)->CPU(%lld)\n", res->mgpu, res->mcpu);
#endif
        hipMemcpy(res->waveform, res->waveform_gpu, n * sizeof(Float64), hipMemcpyDeviceToHost);
        hipMemcpy(res->shift, res->shift_gpu, n * sizeof(Int64), hipMemcpyDeviceToHost);
        hipMemcpy(res->polarity, res->polarity_gpu, n * sizeof(Float64), hipMemcpyDeviceToHost);
        hipMemcpy(res->ps_ratio, res->ps_ratio_gpu, n * sizeof(Float64), hipMemcpyDeviceToHost);
#ifdef DEBUG
        hipMemcpy(&fbuf, res->waveform_gpu, sizeof(Float64), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("(Result_xPU_sync) waveform[0] cpu: %lf, gpu: %lf\n", res->waveform[0], fbuf);
#endif
    }
    res->mcpu = 0;
    res->mgpu = 0;
    hipDeviceSynchronize();
}

void Result_xPU_read(Result_xPU *res, FILE *fp)
{
    fread(&res->n_freq, sizeof(Int64), 1, fp);
    fread(&res->n_phase, sizeof(Int64), 1, fp);
    fread(&res->n_fm, sizeof(Int64), 1, fp);
    Int64 n = res->n_freq * res->n_phase * res->n_fm;
    Result_xPU_alloc(res, res->n_freq, res->n_phase, res->n_fm);
    fread(res->waveform, sizeof(Float64), n, fp);
    fread(res->shift, sizeof(Int64), n, fp);
    fread(res->polarity, sizeof(Float64), n, fp);
    fread(res->ps_ratio, sizeof(Float64), n, fp);
    res->mcpu = _max_(res->mcpu, res->mgpu) + 1;
    Result_xPU_sync(res);
}

void Result_xPU_write(Result_xPU *res, FILE *fp)
{
    Result_xPU_sync(res);
    Int64 n = res->n_freq * res->n_phase * res->n_fm;
    fwrite(&res->n_freq, sizeof(Int64), 1, fp);
    fwrite(&res->n_phase, sizeof(Int64), 1, fp);
    fwrite(&res->n_fm, sizeof(Int64), 1, fp);
    fwrite(res->waveform, sizeof(Float64), n, fp);
    fwrite(res->shift, sizeof(Int64), n, fp);
    fwrite(res->polarity, sizeof(Float64), n, fp);
    fwrite(res->ps_ratio, sizeof(Float64), n, fp);
}
